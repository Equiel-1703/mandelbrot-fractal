#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>

// Include OpenMP for measuring execution time
#include <omp.h>

#include "BMP.hpp"

#define MAX_ITERATIONS 1'000
#define THRESHOLD 2.0

__device__ int calculateMandelbrot(double real, double img)
{
    std::complex<double> c(real, img);
    std::complex<double> z(0.0, 0.0);

    int i = 1;
    while (i < MAX_ITERATIONS)
    {
        z = z * z + c;

        if (std::abs(z) > THRESHOLD)
            break;

        ++i;
    }

    return i;
}

__global__ void createFractal(int *fractals, int *img_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int IMG_SIZE = *img_size;

    if (x >= IMG_SIZE || y >= IMG_SIZE)
        return;

    const double x_min = -2.0, x_max = 2.0;
    const double y_min = -2.0, y_max = 2.0;
    const double scale = 0.5;

    double x_frac, y_frac;

    x_frac = x_min + (x_max - x_min) * (double(x) / double(IMG_SIZE));
    y_frac = y_min + (y_max - y_min) * (double(y) / double(IMG_SIZE));

    x_frac *= scale;
    y_frac *= scale;

    fractals[y * IMG_SIZE + x] = calculateMandelbrot(x_frac, y_frac);
}

int main(int argc, char const *argv[])
{
    if (argc != 2)
    {
        std::cerr << "Usage: " << argv[0] << " <image_size>" << std::endl;
        return 1;
    }

    const int IMG_SIZE = std::stoi(argv[1]);
    if (IMG_SIZE <= 0)
    {
        std::cerr << "ERROR: Image size must be a positive integer." << std::endl;
        return 1;
    }

    if (IMG_SIZE > 100'000)
    {
        std::cerr << "ERROR: Image size is too large. Maximum size is 100,000." << std::endl;
        return 1;
    }

    // Pointers to memory in GPU
    int *fractals, *img_size;
    hipError_t err;

    err = hipMalloc(&fractals, IMG_SIZE * IMG_SIZE * sizeof(int));
    if (err != hipSuccess)
    {
        std::cerr << "ERROR: Memory allocation failed on GPU." << std::endl;
        return 1;
    }

    err = hipMalloc(&img_size, sizeof(int));
    if (err != hipSuccess)
    {
        std::cerr << "ERROR: Memory allocation failed on GPU." << std::endl;
        hipFree(fractals);
        return 1;
    }

    // Copy image size to GPU
    hipMemcpy(img_size, &IMG_SIZE, sizeof(int), hipMemcpyHostToDevice);

    // Get supported device properties
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, 0);
    std::cout << "== Device name: " << deviceProp.name << std::endl;
    std::cout << "== Maximum threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    std::cout << "== Maximum blocks: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << std::endl;

    // Calculate the number of threads and blocks
    int threads_num = sqrt(deviceProp.maxThreadsPerBlock);
    int blocks_num = ceil(double(IMG_SIZE) / double(threads_num));
    
    dim3 threads(threads_num, threads_num, 1);
    dim3 blocks(blocks_num, blocks_num, 1);
    
    std::cout << "== Using blocks: " << blocks.x << " x " << blocks.y << std::endl;
    std::cout << "== Using threads: " << threads.x << " x " << threads.y << std::endl;

    // Start the timer
    double start_time = omp_get_wtime();

    // Launch kernel to create fractal
    createFractal<<<blocks, threads>>>(fractals, img_size);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Stop the timer
    double end_time = omp_get_wtime();

    // Check for errors in kernel launch
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "ERROR: Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(fractals);
        hipFree(img_size);
        return 1;
    }

    // Copy fractal data back to CPU
    int *fractals_cpu = new int[IMG_SIZE * IMG_SIZE];
    err = hipMemcpy(fractals_cpu, fractals, IMG_SIZE * IMG_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "ERROR: Memory copy failed from GPU to CPU." << std::endl;
        delete[] fractals_cpu;
        hipFree(fractals);
        hipFree(img_size);
        return 1;
    }
    // Free GPU memory
    hipFree(fractals);
    hipFree(img_size);
    fractals = nullptr;
    img_size = nullptr;

    // Output the image (Sequential section)
    BMP bmp_image("mandelbrot.bmp", IMG_SIZE, IMG_SIZE);

    for (int x = 0; x < IMG_SIZE; x++)
    {
        for (int y = 0; y < IMG_SIZE; y++)
        {
            int frac_val = fractals_cpu[y * IMG_SIZE + x];

            if (frac_val == MAX_ITERATIONS)
                bmp_image.writePixel(x, y, 0, 0, 0); // É parte do mandelbrot, pintamos de preto
            else
                bmp_image.writePixel(x, y, frac_val % 256, frac_val % 256 / 2, 0);
        }
    }

    bmp_image.save();

    std::cout << std::endl
              << "== Image saved successfully. ==\n";
    std::cout << "Fractal generated: mandelbrot.bmp\n";
    std::cout << "Image size: " << IMG_SIZE << "x" << IMG_SIZE << "\n";
    std::cout << "Max iterations: " << MAX_ITERATIONS << "\n";
    std::cout << "Threshold: " << THRESHOLD << "\n";
    std::cout << "Mandelbrot fractal generated successfully.\n\n";
    std::cout << "== Execution time: " << (end_time - start_time) << " seconds." << std::endl
              << std::endl;

    // Free CPU memory
    delete[] fractals_cpu;
    std::cout << "== Memory freed successfully. ==" << std::endl;
    return 0;
}
